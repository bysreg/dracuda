#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudaScene.hpp"
#include "raytracer_cuda.hpp"
#include "hip/hip_vector_types.h"
#define EPS 0.0001

inline __host__ __device__ float3 quaternionXvector(float4 q, float3 vec)
{
	float3 qvec = make_float3(q.x, q.y, q.z);
	float3 uv = cross(qvec, vec);
	float3 uuv = cross(qvec, uv);
	uv *= (2.0 * q.w);
	uuv *= 2.0;
	return vec + uv + uuv;
}

inline __host__ __device__ float4 quaternionConjugate(float4 q)
{
	return make_float4(-q.x, -q.y, -q.z, q.w);
}

__constant__ cudaScene cuScene;

__device__ float intersectionTest(int type, float3 ray_d, float3 ray_e, int geom)
{
	if (type == 1) {
		float A = dot(ray_d, ray_d);
		float B = dot(ray_d, ray_e);
		float C = dot(ray_e, ray_e) - 1;
		float B24AC = B * B - A * C;
		if (B24AC >= 0) {
			float SB24AC = sqrt(B24AC);
			return (-B - SB24AC) / A;
		}
		return -1;
	} else if (type == 2) {
		float3 v0 = ((float3 *)cuScene.vertex0)[geom];
		float3 v1 = ((float3 *)cuScene.vertex1)[geom];
		float3 v2 = ((float3 *)cuScene.vertex2)[geom];
		float3 t1 = cross(v0 - v2, ray_d);
		float3 t2 = cross(v0 - v1, v0 - ray_e);
		float detA = dot((v0 - v1) ,  t1);
		float distance = dot(v2 - v0, t2) / detA;
		if (distance < EPS)
			return -1;
		float beta = dot(v0 - ray_e, t1) / detA;
		if (beta < 0)
			return -1;
		float gamma = dot(ray_d , t2) / detA;
		if (gamma >= 0 && (beta + gamma) <= 1)
			return distance;
		return -1;
	}
	return -1;
}

__global__
void cudaRayTraceKernel (unsigned char *img)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int w = y * cuScene.width + x;

	img[4 * w + 0] = 255;
	img[4 * w + 1] = 0;
	img[4 * w + 2] = 0;
	img[4 * w + 3] = 0;

	// Calc Ray
	float3 dir = quaternionXvector(*((float4 *)cuScene.cam_orientation), make_float3(0, 0, -1));
	float3 up = quaternionXvector(*((float4 *)cuScene.cam_orientation), make_float3(0, 1, 0));
	float AR = cuScene.aspect;
	float3 cR = cross(dir, up);
	float3 cU = cross(cR, dir);
	float dist = tan(cuScene.fov / 2.0);
	float di = (x + 0.5) / cuScene.width * 2 - 1;
	float dj = (y + 0.5) / cuScene.height * 2 - 1;
	float3 ray_d = normalize(dir + dist * (dj * cU + di * AR * cR));
	float3 ray_e = *((float3 *) cuScene.cam_position);

	float3 *pos_ptr = (float3 *)cuScene.position;
	float4 *rot_ptr = (float4 *)cuScene.rotation;
	float3 *scl_ptr = (float3 *)cuScene.scale;
	int geom = -1;
	float tmin = 10000.0;

	for (int i = 0; i < cuScene.N; i++) {
		float3 t_ray_d = ray_d;
		float3 t_ray_e = ray_e - pos_ptr[i];
		t_ray_d = quaternionXvector(quaternionConjugate(rot_ptr[i]), t_ray_d);
		t_ray_e = quaternionXvector(quaternionConjugate(rot_ptr[i]), t_ray_e);
		t_ray_d = t_ray_d / scl_ptr[i];
		t_ray_e = t_ray_e / scl_ptr[i];
		// Intersection test
		float t = intersectionTest(cuScene.type[i], t_ray_d, t_ray_e, i);
		if (t > EPS && t < tmin) {
			geom = i;
			tmin = t;
		}
	}
	float3 hit = tmin * ray_d + ray_e - pos_ptr[geom];
	hit = quaternionXvector(quaternionConjugate(rot_ptr[geom]), hit) / scl_ptr[geom];
	int type = cuScene.type[geom];
	float3 color = make_float3(0, 0, 0);
	float3 normal;
	// Calc normal
	if (type == 1) {
		normal = hit;
	} else if (type == 2) {
		float3 v0 = ((float3 *)cuScene.vertex0)[geom];
		float3 v1 = ((float3 *)cuScene.vertex1)[geom];
		float3 v2 = ((float3 *)cuScene.vertex2)[geom];
		normal = cross(v1 - v0, v2 - v0);
	}
	// Normal matrix
	normal = normal / scl_ptr[geom];
	normal = quaternionXvector(rot_ptr[geom], normal);
	normal = normalize(normal);


	for (int i = 0; i < cuScene.N_light; i++) {
		float3 diffuse = ((float3 *)cuScene.diffuse)[cuScene.material[geom]];
		float3 light_pos = ((float3 *)cuScene.light_pos)[i];
		float3 light_dir = normalize(light_pos - hit); 
		float cos_factor = dot(light_dir, normal);
		if (cos_factor > 0)
			color += diffuse * cos_factor;
	}
	img[4 * w + 0] = color.x * 255;
	img[4 * w + 1] = color.y * 255;
	img[4 * w + 2] = color.z * 255;
	img[4 * w + 3] = 255;
}

void cudaRayTrace(cudaScene *scene, unsigned char *img)
{
	printf("%p\n", img);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuScene), scene, sizeof(cudaScene)));
	dim3 dimBlock(16, 16);
	dim3 dimGrid(scene->width / 16, scene->height / 16);
	cudaRayTraceKernel<<<dimGrid, dimBlock>>>(img);
}

void helloInvoke()
{
	int a[64];
	int b[64];
	int *dev_a;
	int *dev_b;
	for (int i = 0; i < 64; i++)
		b[i] = i;
	gpuErrchk(hipMalloc((void **)&dev_a, sizeof(int) * 64));
	gpuErrchk(hipMalloc((void **)&dev_b, sizeof(int) * 64));
	gpuErrchk(hipMemcpy(dev_b, b, sizeof(int) * 64, hipMemcpyHostToDevice));
	dim3 dimBlock(64);
	dim3 dimGrid(1);
	//hello<<<dimGrid, dimBlock>>>(dev_a, dev_b);
	gpuErrchk(hipMemcpy(a, dev_a, sizeof(int) * 64, hipMemcpyDeviceToHost));
	hipFree(dev_a);
	hipFree(dev_b);
	for (int i = 0; i < 64; i++)
		printf("%d\n", a[i]);
	return;
	
}
